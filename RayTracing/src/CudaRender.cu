#include "hip/hip_runtime.h"
#include "CudaRender.h"
#include "vec3.h"
#include <stdio.h>


typedef uint32_t u32;
__global__ void renderKernel(u32* device_image_data, u32 width, u32 height, vec3 cameraPosition, glm::mat4 inverseProjection, glm::mat4 inverseView) {

    //printf("From cuda code\n");
    u32 x = threadIdx.x + blockIdx.x * blockDim.x;
    u32 y = threadIdx.y + blockIdx.y * blockDim.y;
    if ((x >= width) || (y>= height)) return;

    //camera code 
    glm::vec2 coord = { (float)x / (float)width, (float)y / (float)height };
    coord = coord * 2.0f - 1.0f; // -1 -> 1
    glm::vec4 target = inverseProjection * glm::vec4(coord.x, coord.y, 1, 1);
    glm::vec3 rayDirection = glm::vec3(inverseView * glm::vec4(glm::normalize(glm::vec3(target) / target.w), 0)); // World space

    Ray ray; 
    ray.Origin = cameraPosition;
    ray.Direction = rayDirection;
    Color color = CudaRender::TraceRay(ray);

    color = color.Clamp(0, 1);
    device_image_data[x + y * width] = color.ConvertToRGBA();
    //if(coord[1] > 0.5f) device_image_data[x + y * width] = 0xffffffff;
}

hipError_t addWithCuda(u32* out, u32 width, u32 height, const Camera& camera);

void CudaRender::Render(uint32_t width, uint32_t height, uint32_t* host_image_data, const Camera& camera) {
	addWithCuda(host_image_data, width, height, camera);
}

__device__ Color CudaRender::TraceRay(const Ray& ray)
{
	float radius = 0.5f;
	// rayDirection = glm::normalize(rayDirection);

	// (bx^2 + by^2)t^2 + (2(axbx + ayby))t + (ax^2 + ay^2 - r^2) = 0
	// where
	// a = ray origin
	// b = ray direction
	// r = radius
	// t = hit distance

    float a = glm::dot(ray.Direction, ray.Direction);
    float b = 2.0f * glm::dot(ray.Origin, ray.Direction);
    float c = glm::dot(ray.Origin, ray.Origin) - radius * radius;

	// Quadratic forumula discriminant:
	// b^2 - 4ac

	float discriminant = b * b - 4.0f * a * c;
    if (discriminant < 0.0f)
        return Color(0, 0, 0, 1);

    // Quadratic formula:
    // (-b +- sqrt(discriminant)) / 2a

    float closestT = (-b - sqrt(discriminant)) / (2.0f * a);
    float t0 = (-b + sqrt(discriminant)) / (2.0f * a); // Second hit distance (currently unused)

    glm::vec3 hitPoint = ray.Origin + ray.Direction * closestT;
    vec3 normal = glm::normalize(hitPoint);

    vec3 lightDir = glm::normalize(vec3(-1, -0, -1));
    float lightIntensity = std::fmax(0.0f, dot(normal, -lightDir)); // == cos(angle)

    vec3 sphereColor(1, 0, 1);
    sphereColor *= lightIntensity;

    return Color(sphereColor.r, sphereColor.g, sphereColor.b, 1.0f);
}

//writes to out array using cuda
hipError_t addWithCuda(u32* out, u32 width, u32 height, const Camera& camera)
{
    u32 problem_size = width * height;
    uint32_t* dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, problem_size * sizeof(uint32_t));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    //cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    //if (cudaStatus != hipSuccess) {
    //    fprintf(stderr, "hipMalloc failed!");
    //    goto Error;
    //}

    //cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    //if (cudaStatus != hipSuccess) {
    //    fprintf(stderr, "hipMalloc failed!");
    //    goto Error;
    //}

    //// Copy input vectors from host memory to GPU buffers.
    //cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    //if (cudaStatus != hipSuccess) {
    //    fprintf(stderr, "hipMemcpy failed!");
    //    goto Error;
    //}

    // Launch a kernel on the GPU with one thread for each element.
    int blockSize = 1024;
    int numBlocks = (problem_size + blockSize - 1) / blockSize;
    printf("Num blocks: %d\n", numBlocks);
    int tx = 8;
    int ty = 8;
    // Render our buffer
    dim3 blocks(width / tx, height/ ty);
    dim3 threads(tx, ty);
    auto inverseProjection = camera.GetInverseProjection();
    auto inverseView = camera.GetInverseView();
    auto cameraPos = camera.GetPosition();
    renderKernel << <blocks, threads>> > (dev_c, width, height, cameraPos, inverseProjection, inverseView);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(out, dev_c, problem_size * sizeof(u32), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);

    return cudaStatus;
}
